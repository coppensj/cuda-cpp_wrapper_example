#include "hip/hip_runtime.h"
#include "cuda_example.h"

__device__ float device_function(const float *data, const int row, const int col, const int ncols){
    return data[col + row * ncols] + 1;
}

__global__ void kernel(
        float *result, 
        const float *data,
        const float *params,
        const int nrows,
        const int ncols, 
        const int numparams)
{  
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (col<0 || col>=ncols || row<0 || row>=nrows) return;
    
    result[col + row * ncols] = device_function(data, row, col, ncols);
    result[col + row * ncols] *= params[0] + params[1];
}

void run_cuda_kernel(
        float *result, 
        const float *data,
        const float *params,
        const int nrows,
        const int ncols, 
        const int numparams)
{
	float *d_result, *d_data, *d_params;

	// Allocate GPU memory
	hipMalloc((void**)&d_result, sizeof(float) * nrows * ncols);
	hipMalloc((void**)&d_data, sizeof(float) * nrows * ncols);
	hipMalloc((void**)&d_params, sizeof(float) * numparams);

	// Transfer data from host to device
    hipMemcpy(d_data, data, sizeof(float) * nrows * ncols, hipMemcpyHostToDevice);
    hipMemcpy(d_params, params, sizeof(float) * numparams, hipMemcpyHostToDevice);

    // Configure threads and run kernel
	dim3 block_size(32,32);
    dim3 grid_size((int)((ncols)/32+1), (int)((nrows)/32+1));
	kernel<<<grid_size,block_size>>>(d_result, d_data, d_params, nrows, ncols, numparams);

	// Transfer data back to host memory
    hipMemcpy(result, d_result, sizeof(float) * nrows * ncols, hipMemcpyDeviceToHost);
	
    // Deallocate device memory
    hipFree(d_result);
    hipFree(d_data);
    hipFree(d_params);
}
